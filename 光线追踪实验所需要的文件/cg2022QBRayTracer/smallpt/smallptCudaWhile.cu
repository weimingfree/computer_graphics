#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <hiprand/hiprand_kernel.h>
#include <hip/device_functions.h>

#include "cutil_math.h"

// limited version of checkCudaErrors from hip/hip_runtime_api.h in CUDA examples
#define checkCudaErrors(val) cudaCheck( (val), #val, __FILE__, __LINE__ )

void cudaCheck(hipError_t result, char const *const func, const char *const file, int const line)
{
	if (result) {
		const char *errorName, *errorString;

		errorName = hipGetErrorName(result);
		errorString = hipGetErrorString(result);

		// Make sure we call CUDA Device Reset before exiting
		hipDeviceReset();

		system("Pause");

	}
}

#define M_PI 3.14159265359f  
#define width 512  
#define height 384 
#define samples 1024

struct Ray {
	float3 origin;
	float3 direction;
	__device__ Ray() {}
	__device__ Ray(float3 o, float3 d) : origin(o), direction(d) {}
};

enum Material { Diffuse, Specular, Refraction };

struct Sphere {

	float radius;
	float3 position, emission, color;
	Material material;

	__device__ float intersect_sphere(const Ray &r) const {

		float3 op = r.origin - position;
		float t, epsilon = 0.0001f;           // epsilon required to prevent floating point precision artefacts
		float b = dot(op, r.direction);       // b in quadratic equation
											  // discriminant quadratic equation
		float discriminant = b * b - dot(op, op) + radius * radius;
		                                      // if (disc < 0) no real solution (not interested in complex roots) 
		if (discriminant < 0)  return 0;      // else check for solutions using negative and positive discriminant
		else discriminant = sqrtf(discriminant);

		t = -b - discriminant;                // pick closest point in front of ray origin
		if (t > epsilon)	return t;
		else	{
			t = -b + discriminant;
			if (t > epsilon)	return t;
			else				return 0;
		}
	}
};

// Scene
__constant__ Sphere spheres[] = {
	{ 1e5f,{ 1e5f + 1.0f,   40.8f, 81.6f },{ 0.0f, 0.0f, 0.0f },{ 0.75f, 0.25f, 0.25f }, Diffuse },   // Left 
	{ 1e5f,{ -1e5f + 99.0f, 40.8f, 81.6f },{ 0.0f, 0.0f, 0.0f },{ 0.25f, 0.25f, 0.75f }, Diffuse },   // Right 
	{ 1e5f,{         50.0f, 40.8f,  1e5f },{ 0.0f, 0.0f, 0.0f },{ 0.75f, 0.75f, 0.75f }, Diffuse },   // Back 
	{ 1e5f,{ 50.0f, 40.8f, -1e5f + 600.0f},{ 0.0f, 0.0f, 0.0f },{ 1.00f, 1.00f, 1.00f }, Diffuse },   // Front 
	{ 1e5f,{ 50.0f,  1e5f,         81.6f },{ 0.0f, 0.0f, 0.0f },{ 0.75f, 0.75f, 0.75f }, Diffuse },   // Bottom 
	{ 1e5f,{ 50.0f, -1e5f + 81.6f, 81.6f },{ 0.0f, 0.0f, 0.0f },{ 0.75f, 0.75f, 0.75f }, Diffuse },   //Top 
	{ 16.5f,{ 27.0f, 16.5f, 47.0f },{ 0.0f, 0.0f, 0.0f },{ 1.0f, 1.0f, 1.0f },   Specular }, // small glass sphere 1
	{ 16.5f,{ 73.0f, 16.5f, 78.0f },{ 0.0f, 0.0f, 0.0f },{ 1.0f, 1.0f, 1.0f }, Refraction }, // small glass sphere 2
    { 10.5f,{ 50.0f, 46.5f, 90.0f}, { 0.0f, 0.0f, 0.0f },{ 1.0f, 1.0f, 1.0f }, Diffuse },    // small white sphere 3
	{ 600.0f,{ 50.0f, 681.6f - .77f, 81.6f },{ 2.0f, 1.8f, 1.6f },{ 0.0f, 0.0f, 0.0f }, Diffuse }  // Light
};

__constant__ const int nsphere = sizeof(spheres) / sizeof(Sphere);

__device__ float rgbToLuminance(const float3& rgb)
{
	const float cWeight[3] = { 0.212671f, 0.715160f, 0.072169f };
	return cWeight[0] * rgb.x + cWeight[1] * rgb.y + cWeight[2] * rgb.z;
}

__device__ inline bool intersect_scene(const Ray &r, float &t, int &id, Sphere* sheres, int &nsp)
{
	// t is distance to closest intersection, initialise t to a huge number outside scene
	float d, inf = t = 1e20;

	for (int i = nsp; i--;)
		if ((d = spheres[i].intersect_sphere(r)) && d < t) {
			t = d;
			id = i;
		}
	// returns true if an intersection with the scene occurred, false when no hit
	return t < inf;
}

inline __host__ __device__ float clamp(float x) { return x < 0 ? 0 : x>1 ? 1 : x; }

inline __host__ __device__ int toInt(float x) { return int(pow(clamp(x), 1 / 2.2) * 255 + 0.5); }

__device__ float gammaCorrection(float x)
{
	return pow(clamp(x), 1 / 2.2f);
}
__device__ inline void maskWeight(float3& mask, float3 color, float cosine, float scale)
{
	mask *= color;
//	mask *= cosine;
//	mask *= scale;
}

__device__ float3 loopRadiance(Ray &r, hiprandState* rs, Sphere* pshere, int &nsp)
{
	// accumulates ray colour with each iteration through bounce loop
	int depth = 0;
	float3 mask = make_float3(1.0f, 1.0f, 1.0f);
	float3 accucolor = make_float3(0.0f, 0.0f, 0.0f);

	// ray bounce loop
	while (true) {
		float t;
		int id = 0;

		// find closest intersection with object's index, if miss break.
		if (!intersect_scene(r, t, id, pshere, nsp))	break;

		const Sphere &obj = spheres[id];
		float3 x = r.origin + r.direction*t;                 // hitpoint
		float3 n = normalize(x - obj.position);              // normal
		float3 nl = dot(n, r.direction) < 0 ? n : n * -1;    // front facing normal
														 
		r.origin = x + nl * 0.05f;                           // prevent self-intersection	
		accucolor += mask * obj.emission;                    // emissive

		if (obj.material == Diffuse) {    //diffuse
            // uniform sampling hemisphere
			float r1 = 2 * M_PI * hiprand_uniform(rs);
			float r2 = hiprand_uniform(rs);
			float r2s = sqrtf(r2);

			// compute local coordinate on the hit point
			float3 w = nl;
			float3 u = normalize(cross((fabs(w.x) > .1 ? make_float3(0, 1, 0) : make_float3(1, 0, 0)), w));
			float3 v = cross(w, u);

			// local to world convert
			r.origin = x + nl * 0.05f;              // offset for self intersection
			r.direction = normalize(u*cos(r1)*r2s + v * sin(r1)*r2s + w * sqrtf(1 - r2));
            // weigh light contribution using cosine of angle between
			// incident light and normal fudge factor
			maskWeight(mask, obj.color, dot(r.direction, nl), 2);
		} else if (obj.material == Specular) {  //specular
			r.direction = r.direction - n * 2 * dot(n, r.direction);
			r.origin = x + r.direction * 0.07f;
			maskWeight(mask, obj.color, dot(r.direction, nl), 2);
		} else { //refraction
			r.origin = x;
			bool into = dot(n, nl) > 0;                 // Ray from outside going in?
			float nc = 1, nt = 1.5, nnt = into ? nc / nt : nt / nc, ddn = dot(r.direction, nl), cos2t;
			// Ideal dielectric REFRACTION
			float3 reflectDir = r.direction - n * 2 * dot(n, r.direction);
			
			// total internal reflection
			if ((cos2t = 1 - nnt*nnt*(1 - ddn*ddn)) < 0) {
				r.direction = reflectDir;
				maskWeight(mask, obj.color, dot(r.direction, nl), 2);
			} else {  // refract or reflect
				float3 tdir = normalize(r.direction*nnt - n*((into ? 1 : -1)*(ddn*nnt + sqrt(cos2t))));
				float a = nt - nc, b = nt + nc, R0 = a*a / (b*b), c = 1 - (into ? -ddn : dot(tdir, n));
				float Re = R0 + (1 - R0)*c*c*c*c*c, Tr = 1 - Re, P = .25 + .5*Re, RP = Re / P, TP = Tr / (1 - P);
			    if (hiprand_uniform(rs) < P)	{  // reflect		
				  r.direction = reflectDir;
				  maskWeight(mask, obj.color, dot(r.direction, nl), 2);
				  mask *= RP;
			    } else {		               // refract
				r.direction = tdir;
				maskWeight(mask, obj.color, dot(r.direction, nl), 2);
				mask *= TP;
			}
		  }
		}
		// Russian roulette Stop with at least some probability to avoid getting stuck
		if (depth++ >= 5) {
			float q = min(0.95f, rgbToLuminance(mask));
			if (hiprand_uniform(rs) >= q)
				break;
			mask /= q;
		}
	}
	return accucolor;
}

__global__ void smallptRenderkernel(float3 *output, unsigned int outputSize)
{
	//copy spheres to shared memory
	__shared__ int nsp;
	__shared__ Sphere sspheres[nsphere];
	__shared__ Ray tRay;

	nsp = nsphere;
	sspheres[threadIdx.x % nsp] = spheres[threadIdx.x % nsp];

	__syncthreads();

	// position of current pixel
	unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
	unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;

	if ((x >= width) || (y >= height)) return;

	// index of current pixel
	//int i = (blockIdx.x + blockIdx.y * gridDim.x) * (blockDim.x * blockDim.y) + (threadIdx.y * blockDim.x) + threadIdx.x;
	unsigned int i = (height - y - 1)*width + x;

	if (i >= outputSize) return;

	hiprandState rs;
	hiprand_init(i, 0, 0, &rs);

	float3 r = make_float3(0.0f);
	Ray cam(make_float3(50.0, 52.0, 295.6), normalize(make_float3(0.0, -0.042612, -1)));
	float3 cx = make_float3(width * 0.5135 / height, 0.0f, 0.0f);
	float3 cy = normalize(cross(cx, cam.direction)) * 0.5135;

	for (int sy = 0; sy < 2; sy++)	{
		for (int sx = 0; sx < 2; sx++)	{

			for (int s = 0; s < samples; s++) {
				float r1 = hiprand_uniform(&rs);
				float dx = r1 < 1 ? sqrtf(r1) - 1 : 1 - sqrtf(2 - r1);
				float r2 = hiprand_uniform(&rs);
				float dy = r2 < 1 ? sqrtf(r2) - 1 : 1 - sqrtf(2 - r2);
				//--! super sampling
				float3 d = cam.direction + cx*((((sx + dx + .5) / 2) + x) / width - .5) +
					                       cy*((((sy + dy + .5) / 2) + y) / height - .5);

				//Ray tRay = Ray(cam.origin + d * 140, normalize(d));
				tRay.direction = normalize(d);
				tRay.origin = cam.origin + d * 40;
				r += loopRadiance(tRay, &rs, sspheres, nsp) *(.25f / samples);
			}
		}
	}	
	// output to the cache
	__shared__ float3 temp;
	temp = make_float3(clamp(r.x, 0.0f, 1.0f), clamp(r.y, 0.0f, 1.0f), clamp(r.z, 0.0f, 1.0f));
	output[i] = temp;
}

extern "C"  int smallptRayTrace(unsigned int *sample)
{

	// get number of SMs on this GPU
//	int devID = 0;
//	hipDeviceProp_t deviceProps;
//	checkCudaErrors(hipGetDeviceProperties(&deviceProps, devID));

	hipDeviceReset();

	hipError_t cudaStatus;
	// Choose which GPU to run on
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) return 0;

	float3* output_d;
	float3* output_h = new float3[width*height];
	
	// Allocate GPU buffers for outout image
	cudaStatus = hipMalloc(&output_d, width * height * sizeof(float3));
	if (cudaStatus != hipSuccess) return 0;

	dim3 block(8, 8, 1);
	dim3 grid(width / block.x, height / block.y, 1);

	// Launch a kernel on the GPU with one thread for each element.
	smallptRenderkernel<<< grid, block >>>(output_d, width * height);

	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) return 0;

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) return 0;

	// Copy output vector from GPU buffer to host memory.
	cudaStatus = hipMemcpy(output_h, output_d, width * height * sizeof(float3), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) return 0;

	for (int i = 0; i < width*height; i++) {
		unsigned int red = 0;
		unsigned int green = 0;
		unsigned int blue = 0;

		  red   = toInt(output_h[i].x);
		  green = toInt(output_h[i].y);
		  blue  = toInt(output_h[i].z);

		  sample[i] = (0xFF << 24) | (red << 16) | (green << 8) | blue;
	}

	delete[] output_h;
	hipFree(output_d);

	return 1;
}
