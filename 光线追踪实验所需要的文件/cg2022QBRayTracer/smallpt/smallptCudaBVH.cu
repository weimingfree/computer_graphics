#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <hip/hip_runtime.h>
#include <driver_functions.h>

#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <hiprand/hiprand_kernel.h>
#include <hip/device_functions.h>

#include "smallptCudaBVH.h"

#define ARRAY_SIZE(array) (sizeof((array))/sizeof((array[0])))
#define BLOCK_SIZE 256
#define CODE_OFFSET (1<<21)
#define CODE_LENGTH (21)
#define INTERSECT_STACK_SIZE (18)
#define RESTRUCT_STACK_SIZE (4)
#define Ci 1.2
#define Cl 0.0
#define Ct 1.0

// limited version of checkCudaErrors from hip/hip_runtime_api.h in CUDA examples
#define checkCudaErrors(val) cudaCheck( (val), #val, __FILE__, __LINE__ )

void cudaCheck(hipError_t result, char const *const func, const char *const file, int const line)
{
	if (result) {
		const char *errorName, *errorString;

		errorName = hipGetErrorName(result);
		errorString = hipGetErrorString(result);

		// Make sure we call CUDA Device Reset before exiting
		hipDeviceReset();

		system("Pause");

	}
}

#define M_PI 3.14159265359f  

inline __host__ __device__ double clamp(double x) { return x<0 ? 0 : x>1 ? 1 : x; }

inline __host__ __device__ int toInt(double x) { return int(pow(clamp(x), 1 / 2.2) * 255 + .5); }

__device__ double drandom(hiprandState *s)
{
	double d = hiprand_uniform_double(s);
	return d;
}

inline __device__ float min2(float a, float b)
{
	return (a < b) ? a : b;
}

inline __device__ float max2(float a, float b)
{
	return (a > b) ? a : b;
}

__device__ void merge_bounds(Bound& b1, Bound& b2, Bound* b3)
{
	b3->min_x = min2(b1.min_x, b2.min_x);
	b3->max_x = max2(b1.max_x, b2.max_x);
	b3->min_y = min2(b1.min_y, b2.min_y);
	b3->max_y = max2(b1.max_y, b2.max_y);
	b3->min_z = min2(b1.min_z, b2.min_z);
	b3->max_z = max2(b1.max_z, b2.max_z);
	return;
}

inline __device__ int intMin(int i, int j)
{
	return (i > j) ? j : i;
}

inline __device__ int intMax(int i, int j)
{
	return (i > j) ? i : j;
}

/**
* Longest common prefix for morton code
*/
inline __device__ int longestCommonPrefix(int i, int j, int len)
{
	if (0 <= j && j < len) {
		return __clz(i ^ j);
	}
	else {
		return -1;
	}
}

/**
* Test if a ray intersect a bound
*/
__device__ bool intersection_bound_test(const Ray &r, Bound& bound)
{
	float t_min, t_max, t_xmin, t_xmax, t_ymin, t_ymax, t_zmin, t_zmax;
	float x_a = 1.0 / r.d.x, y_a = 1.0 / r.d.y, z_a = 1.0 / r.d.z;
	float  x_e = r.o.x, y_e = r.o.y, z_e = r.o.z;

	// calculate t interval in x-axis
	if (x_a >= 0) {
		t_xmin = (bound.min_x - x_e) * x_a;
		t_xmax = (bound.max_x - x_e) * x_a;
	}
	else {
		t_xmin = (bound.max_x - x_e) * x_a;
		t_xmax = (bound.min_x - x_e) * x_a;
	}

	// calculate t interval in y-axis
	if (y_a >= 0) {
		t_ymin = (bound.min_y - y_e) * y_a;
		t_ymax = (bound.max_y - y_e) * y_a;
	}
	else {
		t_ymin = (bound.max_y - y_e) * y_a;
		t_ymax = (bound.min_y - y_e) * y_a;
	}

	// calculate t interval in z-axis
	if (z_a >= 0) {
		t_zmin = (bound.min_z - z_e) * z_a;
		t_zmax = (bound.max_z - z_e) * z_a;
	}
	else {
		t_zmin = (bound.max_z - z_e) * z_a;
		t_zmax = (bound.min_z - z_e) * z_a;
	}

	// find if there an intersection among three t intervals
	t_min = max2(t_xmin, max2(t_ymin, t_zmin));
	t_max = min2(t_xmax, min2(t_ymax, t_zmax));

	return (t_min <= t_max);
}

/**
* Intersect test in BVH
*/
__device__ bool intersect(Sphere *start, TreeNode *cur,
	                      const Ray &r, double &t, int &id)
{
	// int n = 9;
	// double d, inf=t=1e20;
	// for(int i=n;i--;) if((d=start[i].intersect(r))&&d<t){t=d;id=i;}
	// return t<inf;

	// Use static allocation because malloc() can't be called in parallel
	// Use stack to traverse BVH to save space (cost is O(height))
	TreeNode *stack[INTERSECT_STACK_SIZE];
	int topIndex = INTERSECT_STACK_SIZE;
	stack[--topIndex] = cur;
	bool intersected = false;

	// Do while stack is not empty
	while (topIndex != INTERSECT_STACK_SIZE) {
		TreeNode *n = stack[topIndex++];
		if (intersection_bound_test(r, n->bound)) {
			if (n->leaf) {
				double d = n->sphere->intersect(r);
				if (d != 0.0) {
					if (d < t) {
						t = d;
						id = n->sphere->index;
					}
					intersected = true;
				}
			}
			else {
				stack[--topIndex] = n->right;
				stack[--topIndex] = n->left;

				if (topIndex < 0) {
//					printf("Intersect stack not big enough. Increase INTERSECT_STACK_SIZE!\n");
					return false;
				}
			}
		}
	}

	return intersected;
}

__device__ Vec radiance(Sphere *start, TreeNode *cur, const Ray &r_,
	                    int depth_, hiprandState *s)
{
	double t;                               // distance to intersection
	int id = 0;                               // id of intersected object
	Ray r = r_;
	int depth = depth_;
	Vec cl(0, 0, 0);   // accumulated color
	Vec cf(1, 1, 1);  // accumulated reflectance
	while (1) {
		t = 1e20;
		if (!intersect(start, cur, r, t, id)) return cl; // if miss, return black
		Sphere &obj = start[id];        // the hit object
		Vec x = r.o + r.d*t, n = (x - obj.p).norm(), nl = n.dot(r.d)<0 ? n : n*-1, f = obj.c;
		double p = f.x>f.y && f.x>f.z ? f.x : f.y>f.z ? f.y : f.z; // max refl
		cl = cl + cf.mult(obj.e);
		if (++depth>5) if (drandom(s)<p) f = f*(1 / p); else return cl; //R.R.
		cf = cf.mult(f);
		if (obj.refl == DIFF) {                  // Ideal DIFFUSE reflection
			double r1 = 2 * M_PI*drandom(s), r2 = drandom(s), r2s = sqrt(r2);
			Vec w = nl, u = ((fabs(w.x)>.1 ? Vec(0, 1) : Vec(1)) % w).norm(), v = w%u;
			Vec d = (u*cos(r1)*r2s + v*sin(r1)*r2s + w*sqrt(1 - r2)).norm();
			r = Ray(x, d);
			continue;
		}
		else if (obj.refl == SPEC) {           // Ideal SPECULAR reflection
			r = Ray(x, r.d - n * 2 * n.dot(r.d));
			continue;
		}
		Ray reflRay(x, r.d - n * 2 * n.dot(r.d));     // Ideal dielectric REFRACTION
		bool into = n.dot(nl)>0;                // Ray from outside going in?
		double nc = 1, nt = 1.5, nnt = into ? nc / nt : nt / nc, ddn = r.d.dot(nl), cos2t;
		if ((cos2t = 1 - nnt*nnt*(1 - ddn*ddn))<0) {    // Total internal reflection
			r = reflRay;
			continue;
		}
		Vec tdir = (r.d*nnt - n*((into ? 1 : -1)*(ddn*nnt + sqrt(cos2t)))).norm();
		double a = nt - nc, b = nt + nc, R0 = a*a / (b*b), c = 1 - (into ? -ddn : tdir.dot(n));
		double Re = R0 + (1 - R0)*c*c*c*c*c, Tr = 1 - Re, P = .25 + .5*Re, RP = Re / P, TP = Tr / (1 - P);
		if (drandom(s)<P) {
			cf = cf*RP;
			r = reflRay;
		}
		else {
			cf = cf*TP;
			r = Ray(x, tdir);
		}
		continue;
	}
}

/**
* Ray trace kernel
* Use BVH for better performance
*/
__global__ void kernelRayTrace(hiprandState* states, Vec *deviceSubpixelBuffer,
	                           int width, int height, int samps, Sphere *start,
	TreeNode *cudaDeviceTreeNodes, Ray cam, Vec cx, Vec cy)
{

	int subpixelIndex = blockIdx.x * blockDim.x + threadIdx.x;
	if (subpixelIndex >= width * height * 4 || subpixelIndex < 0) return;
	int pixelIndex = subpixelIndex / 4;

	int y = pixelIndex / width;
	int x = pixelIndex % width;
	int sy = (subpixelIndex % 4) / 2;
	int sx = (subpixelIndex % 4) % 2;

	if (x < 0 || y < 0 || x >= width || y >= height) {
		return;
	}

	hiprand_init(y*y*y, subpixelIndex, 0, &states[subpixelIndex]);
	hiprandState state = states[subpixelIndex];

	Vec r = Vec();
	for (int s = 0; s<samps; s++) {
		double r1 = 2 * drandom(&state), dx = r1<1 ? sqrt(r1) - 1 : 1 - sqrt(2 - r1);
		double r2 = 2 * drandom(&state), dy = r2<1 ? sqrt(r2) - 1 : 1 - sqrt(2 - r2);
		Vec d = cx*(((sx + .5 + dx) / 2 + x) / width - .5) +
			cy*(((sy + .5 + dy) / 2 + y) / height - .5) + cam.d;
		r = r + radiance(start, cudaDeviceTreeNodes, Ray(cam.o + d * 140, d.norm()), 0, &state) * (1. / samps);
	}

	deviceSubpixelBuffer[subpixelIndex] = r;
}

/**
* Get result kernel
* Combine subpixel colors into one
*/
__global__ void kernelGetResult(Vec *deviceSubpixelBuffer,
	                            Vec *devicePixelBuffer, int width, int height)
{

	int index = blockIdx.x * blockDim.x + threadIdx.x;
	if (index >= width * height) {
		return;
	}

	Vec res = Vec();
	for (int i = 0; i < 4; i++) {
		Vec subpixelVec = deviceSubpixelBuffer[index * 4 + i];
		res = res + Vec(clamp(subpixelVec.x),
			            clamp(subpixelVec.y),
			            clamp(subpixelVec.z)) * .25;
	}

	devicePixelBuffer[index] = res;
}

/**
* Radix tree construction kernel
* Algorithm described in karras2012 paper.
* Node-wise parallel
*/
__global__ void kernelConstructRadixTree(int len, TreeNode *radixTreeNodes,
	                                              TreeNode *radixTreeLeaves)
{

	int i = blockIdx.x * blockDim.x + threadIdx.x;

	if (i >= len) return;

	// Run radix tree construction algorithm
	// Determine direction of the range (+1 or -1)
	int d = longestCommonPrefix(i, i + 1, len + 1) -
		longestCommonPrefix(i, i - 1, len + 1) > 0 ? 1 : -1;

	// Compute upper bound for the length of the range
	int sigMin = longestCommonPrefix(i, i - d, len + 1);
	int lmax = 2;

	while (longestCommonPrefix(i, i + lmax * d, len + 1) > sigMin) {
		lmax *= 2;
	}

	// Find the other end using binary search
	int l = 0;
	int divider = 2;
	for (int t = lmax / divider; t >= 1; divider *= 2) {
		if (longestCommonPrefix(i, i + (l + t) * d, len + 1) > sigMin) {
			l += t;
		}
		t = lmax / divider;
	}

	int j = i + l * d;


	//printf("i:%d d:%d lmax:%d l:%d j:%d \n",i , d, lmax, l, j);
	// Find the split position using binary search
	int sigNode = longestCommonPrefix(i, j, len + 1);
	int s = 0;
	divider = 2;
	for (int t = (l + (divider - 1)) / divider; t >= 1; divider *= 2) {
		if (longestCommonPrefix(i, i + (s + t) * d, len + 1) > sigNode) {
			s = s + t;
		}
		t = (l + (divider - 1)) / divider;
	}

	int gamma = i + s * d + intMin(d, 0);

	// Output child pointers
	TreeNode *current = radixTreeNodes + i;


	if (intMin(i, j) == gamma) {
		current->left = radixTreeLeaves + gamma;
		(radixTreeLeaves + gamma)->parent = current;
	}
	else {
		current->left = radixTreeNodes + gamma;
		(radixTreeNodes + gamma)->parent = current;
	}

	if (intMax(i, j) == gamma + 1) {
		current->right = radixTreeLeaves + gamma + 1;
		(radixTreeLeaves + gamma + 1)->parent = current;
	}
	else {
		current->right = radixTreeNodes + gamma + 1;
		(radixTreeNodes + gamma + 1)->parent = current;
	}

	current->min = intMin(i, j);
	current->max = intMax(i, j);
}

__device__ bool check_bound(TreeNode *p, TreeNode *l, TreeNode *r) 
{
	return (
		p->bound.min_x == min2(l->bound.min_x, r->bound.min_x) &&
		p->bound.max_x == max2(l->bound.max_x, r->bound.max_x) &&
		p->bound.min_y == min2(l->bound.min_y, r->bound.min_y) &&
		p->bound.max_y == max2(l->bound.max_y, r->bound.max_y) &&
		p->bound.min_z == min2(l->bound.min_z, r->bound.min_z) &&
		p->bound.max_z == max2(l->bound.max_z, r->bound.max_z)
		);
}

__device__ bool check_sanity(TreeNode *n)
{
	if (n->leaf) {
		return true;
	}
	else {
		return (
			n->left->parent == n &&
			n->right->parent == n
			);
	}
}

/**
* BVH Construction kernel
* Algorithm described in karras2012 paper (bottom-up approach).
*/
__global__ void kernelConstructBVHTree(int len, TreeNode *treeNodes, TreeNode *treeLeaves,
	                                   int *nodeCounter, int *sorted_geometry_indices, Sphere *spheres)
{

	int index = blockIdx.x * blockDim.x + threadIdx.x;

	if (index >= len) return;

	TreeNode *leaf = treeLeaves + index;

	// Handle leaf first
	int geometry_index = sorted_geometry_indices[index];
	leaf->bound = spheres[geometry_index].bound;
	leaf->sphere = &(spheres[geometry_index]);

	TreeNode *current = leaf->parent;
	int currentIndex = current - treeNodes;
	int res = atomicAdd(nodeCounter + currentIndex, 1);

	// Go up and handle internal nodes
	while (1) {
		if (res == 0) {
			return;
		}

		merge_bounds(current->left->bound, current->right->bound,
			&(current->bound));

		// If current is root, return
		if (current == treeNodes) {
			return;
		}
		current = current->parent;
		currentIndex = current - treeNodes;
		res = atomicAdd(nodeCounter + currentIndex, 1);
	}
}

__inline__ __host__ __device__ float getArea(float min_x, float max_x,
	                                         float min_y, float max_y,
	                                         float min_z, float max_z)
{
	float dx = max_x - min_x;
	float dy = max_y - min_y;
	float dz = max_z - min_z;
	return 2 * (dx * dy + dx * dz + dy * dz);
}

__host__ __device__ int pwr(int base, unsigned exp)
{
	int acc = 1;
	for (unsigned c = 0; c < exp; c++) {
		acc *= base;
	}
	return acc;
}

__host__ __device__ float get_total_area(int n, TreeNode *leaves[], unsigned s)
{
	float lmin_x, lmin_y, lmin_z, lmax_x, lmax_y, lmax_z;
	float min_x = pos_infinity;
	float max_x = neg_infinity;
	float min_y = pos_infinity;
	float max_y = neg_infinity;
	float min_z = pos_infinity;
	float max_z = neg_infinity;
	for (int i = 0; i < n; i++) {
		if ((s >> i) & 1 == 1) {
			lmin_x = leaves[i]->bound.min_x;
			lmin_y = leaves[i]->bound.min_y;
			lmin_z = leaves[i]->bound.min_z;
			lmax_x = leaves[i]->bound.max_x;
			lmax_y = leaves[i]->bound.max_y;
			lmax_z = leaves[i]->bound.max_z;
			if (lmin_x < min_x) min_x = lmin_x;
			if (lmin_y < min_y) min_y = lmin_y;
			if (lmin_z < min_z) min_z = lmin_z;
			if (lmax_x > max_x) max_x = lmax_x;
			if (lmax_y > max_y) max_y = lmax_y;
			if (lmax_z > max_z) max_z = lmax_z;
		}
	}
	return getArea(min_x, max_x, min_y, max_y, min_z, max_z);
}

__host__ __device__ void calculateOptimalTreelet(int n, TreeNode **leaves,
	                                             unsigned char *p_opt)
{
	int num_subsets = pwr(2, n) - 1;
	// 0th element in array should not be used
	float a[128];
	float c_opt[128];
	// Calculate surface area for each subset
	for (unsigned char s = 1; s <= num_subsets; s++) {
		a[s] = get_total_area(n, leaves, s);
	}
	// Initialize costs of individual leaves
	for (unsigned i = 0; i <= (n - 1); i++) {
		c_opt[pwr(2, i)] = leaves[i]->cost;
	}
	// Optimize every subset of leaves
	for (unsigned k = 2; k <= n; k++) {
		for (unsigned char s = 1; s <= num_subsets; s++) {
			if (__popc(s) == k) {
				// Try each way of partitioning the leaves
				float c_s = pos_infinity;
				unsigned char p_s = 0;
				unsigned char d = (s - 1) & s;
				unsigned char p = (-d) & s;
				while (p != 0) {
					float c = c_opt[p] + c_opt[s ^ p];
					if (c < c_s) {
						c_s = c;
						p_s = p;
					}
					//printf("p=%x, c=%.0lf, c_s=%.0lf, p_s=%x\n", p & 0xff, c, c_s, p_s & 0xff);
					p = (p - d) & s;
				}
				// Calculate final SAH cost
				c_opt[s] = Ci * a[s] + c_s;
				p_opt[s] = p_s;
			}
		}
	}
}

__device__ void propagateAreaCost(TreeNode *root, TreeNode **leaves, int num_leaves)
{

	for (int i = 0; i < num_leaves; i++) {
		TreeNode *cur = leaves[i];
		cur = cur->parent;
		while (cur != root) {
			if (cur->cost == 0.0) {
				if (cur->left->cost != 0.0 && cur->right->cost != 0.0) {
					// Both left & right propagated
					Bound *bound = &cur->bound;
					merge_bounds(cur->left->bound, cur->right->bound, bound);
					cur->area = getArea(bound->min_x, bound->max_x, bound->min_y,
						bound->max_y, bound->min_z, bound->max_z);
					cur->cost = Ci * cur->area + cur->left->cost + cur->right->cost;
				}
				else {
					// Only one side propagated
					break;
				}
			}
			cur = cur->parent;
		}
	}

	// Propagate root
	Bound *bound = &root->bound;
	merge_bounds(root->left->bound, root->right->bound, bound);
	root->area = getArea(bound->min_x, bound->max_x, bound->min_y,
		bound->max_y, bound->min_z, bound->max_z);
	root->cost = Ci * root->area + root->left->cost + root->right->cost;
}

struct PartitionEntry {
	unsigned char partition;
	bool left;
	TreeNode *parent;
};

__device__ void restructTree(TreeNode *parent, TreeNode **leaves,
	                         TreeNode **nodes, unsigned char partition, unsigned char *optimal,
	                         int &index, bool left, int num_leaves)
{
	PartitionEntry stack[RESTRUCT_STACK_SIZE];
	int topIndex = RESTRUCT_STACK_SIZE;
	PartitionEntry tmp = { partition, left, parent };
	stack[--topIndex] = tmp;

	// Do while stack is not empty
	while (topIndex != RESTRUCT_STACK_SIZE) {
		PartitionEntry *pe = &stack[topIndex++];
		partition = pe->partition;
		left = pe->left;
		parent = pe->parent;

		if (partition == 0) return;

		if (__popc(partition) == 1) {    // Leaf
			
			int leaf_index = __ffs(partition) - 1;

			TreeNode *leaf = leaves[leaf_index];
			if (left) {
				parent->left = leaf;
			} else {
				parent->right = leaf;
			}
			leaf->parent = parent;
		} else {                        // Internal node
			
			if (index >= 7) return;

			TreeNode *node = nodes[index++];

			// Set cost to 0 as a mark
			node->cost = 0.0;

			if (left) {
				parent->left = node;
			} else {
				parent->right = node;
			}
			node->parent = parent;

			if (partition >= 128) return;

			unsigned char left_partition = optimal[partition];
			unsigned char right_partition = (~left_partition) & partition;

			if ((left_partition | partition) != partition) {
//				printf("left error: %x vs %x\n", left_partition & 0xff, partition & 0xff);
				return;
			}
			if ((right_partition | partition) != partition) {
//				printf("right error: %x vs %x\n", right_partition & 0xff, partition & 0xff);
				return;
			}

			if (topIndex < 2) {
				printf("restructTree stack not big enough. Increase RESTRUCT_STACK_SIZE!\n");
			}
			PartitionEntry tmp1 = { left_partition, true, node };
			stack[--topIndex] = tmp1;
			PartitionEntry tmp2 = { right_partition, false, node };
			stack[--topIndex] = tmp2;
		}
	}

	propagateAreaCost(parent, leaves, num_leaves);
}

__device__ void printPartition(TreeNode *root, unsigned char *optimal,
	                           unsigned char start, unsigned char mask)
{
	int level = 1;
	Queue *q = new Queue();
	q->push((void *)start);
	q->push((void *)((~start) & mask));

	Queue *qt = new Queue();

	while (!q->empty()) {

		while (!q->empty()) {
			unsigned char n = (unsigned char)(unsigned long)(q->last());
			q->pop();

			if (__popc(n) != 1) {
//				printf("[%d %p] %x\n", level, root, n & 0xff);
				qt->push((void *)optimal[n]);
				qt->push((void *)((~optimal[n]) & n));
			}
			else {
//				printf("[%d %p] (%d)\n", level, root, __ffs(n));
			}
		}
		level++;

		Queue *t = q;
		q = qt;
		qt = t;
	}

	delete q;
	delete qt;
}

/**
* treeletOptimize
* Find the treelet and optimize
*/
__device__ void treeletOptimize(TreeNode *root) 
{
	// Don't need to optimize if root is a leaf
	if (root->leaf) return;

	// Find a treelet with max number of leaves being 7
	TreeNode *leaves[7];
	int counter = 0;
	leaves[counter++] = root->left;
	leaves[counter++] = root->right;

	// Also remember the internal nodes
	// Max 7 (leaves) - 1 (root doesn't count) - 1
	TreeNode *nodes[5];
	int nodes_counter = 0;

	float max_area;
	int max_index = 0;

	while (counter < 7 && max_index != -1) {
		max_index = -1;
		max_area = -1.0;

		for (int i = 0; i < counter; i++) {
			if (!(leaves[i]->leaf)) {
				float area = leaves[i]->area;
				if (area > max_area) {
					max_area = area;
					max_index = i;
				}
			}
		}

		if (max_index != -1) {

			TreeNode *tmp = leaves[max_index];

			// Put this node in nodes array
			nodes[nodes_counter++] = tmp;

			// Replace the max node with its children
			leaves[max_index] = leaves[counter - 1];
			leaves[counter - 1] = tmp->left;
			leaves[counter++] = tmp->right;
		}
	}
	/*
#ifdef DEBUG_PRINT
	printf("%p counter=%d nodes_counter=%d\n", root, counter, nodes_counter);
	for (int i = 0; i < counter; i++) {
		printf("%p leaf %p\n", root, leaves[i]);
	}
	for (int i = 0; i < nodes_counter; i++) {
		printf("%p node %p\n", root, nodes[i]);
	}
#endif
	*/

	unsigned char optimal[128];

	// Call calculateOptimalCost here
	calculateOptimalTreelet(counter, leaves, optimal);
/*
#ifdef DEBUG_PRINT
	printPartition(root, optimal, optimal[(1 << counter) - 1], (1 << counter) - 1);
#endif
*/
	// Use complement on right tree, and use original on left tree
	unsigned char mask = (1 << counter) - 1;    // mask = max index
	int index = 0;                              // index for free nodes
	unsigned char leftIndex = mask;
	unsigned char left = optimal[leftIndex];
	restructTree(root, leaves, nodes, left, optimal, index, true, counter);

	unsigned char right = (~left) & mask;
	restructTree(root, leaves, nodes, right, optimal, index, false, counter);

	// Calculate current node's area & cost
	Bound *bound = &root->bound;
	merge_bounds(root->left->bound, root->right->bound, bound);
	root->area = getArea(bound->min_x, bound->max_x, bound->min_y,
		bound->max_y, bound->min_z, bound->max_z);
	root->cost = Ci * root->area + root->left->cost + root->right->cost;
}

/**
* BVH Optimization kernel
*/
__global__ void kernelOptimize(int num_leaves, int *nodeCounter,
	                           TreeNode *treeNodes, TreeNode *treeLeaves)
{

	int index = blockIdx.x * blockDim.x + threadIdx.x;

	if (index >= num_leaves) return;

	TreeNode *leaf = treeLeaves + index;

	// Handle leaf first
	// Leaf's cost is just its bounding volumn's cost
	Bound *bound = &leaf->bound;
	leaf->area = getArea(bound->min_x, bound->max_x, bound->min_y,
		bound->max_y, bound->min_z, bound->max_z);
	leaf->cost = Ct * leaf->area;

	__syncthreads();
/*
#ifdef DEBUG_PRINT
	__syncthreads();
	if (index == 0) {
		printf("Launching Print BVH GPU... (before Optimization)\n");
		printBVH(treeNodes);
		printf("Launched Print BVH GPU... (before Optimization)\n");
	}
	__syncthreads();
#endif
*/
	TreeNode *current = leaf->parent;
	int currentIndex = current - treeNodes;
	int res = atomicAdd(nodeCounter + currentIndex, 1);

	// Go up and handle internal nodes
	while (1) {
		if (res == 0) {
			return;
		}

//		printf("%d Going to optimize %p\n", index, current);

		treeletOptimize(current);

//		printf("%d Optimized %p\n", index, current);

		// If current is root, return
		if (current == treeNodes) {
			return;
		}
		current = current->parent;
		currentIndex = current - treeNodes;
		res = atomicAdd(nodeCounter + currentIndex, 1);
	}

}



extern "C"  int smallptCudaRayTrace(int width, int height, unsigned int *bmpImage, int samples)
{

	// get number of SMs on this GPU
//	int devID = 0;
//	hipDeviceProp_t deviceProps;
//	checkCudaErrors(hipGetDeviceProperties(&deviceProps, devID));

	hipDeviceReset();

	hipError_t cudaStatus;
	// Choose which GPU to run on
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) return 0;

	float3* output_d;
	float3* output_h = new float3[width*height];
	
	// Allocate GPU buffers for outout image
	cudaStatus = hipMalloc(&output_d, width * height * sizeof(float3));
	if (cudaStatus != hipSuccess) return 0;

	dim3 block(8, 8, 1);
	dim3 grid(width / block.x, height / block.y, 1);

	// Launch a kernel on the GPU with one thread for each element.
//	smallptCudakernel <<< grid, block >>>(width, height, output_d, samples);
//	smallptCudakernel <<< height, width >>>(width, height, output_d, samples);

	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) return 0;

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) return 0;

	// Copy output vector from GPU buffer to host memory.
	cudaStatus = hipMemcpy(output_h, output_d, width * height * sizeof(float3), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) return 0;

	for (int i = 0; i < width*height; i++) {
		unsigned int red = 0;
		unsigned int green = 0;
		unsigned int blue = 0;

		  red   = toInt(output_h[i].x);
		  green = toInt(output_h[i].y);
		  blue  = toInt(output_h[i].z);

		  bmpImage[i] = (0xFF << 24) | (red << 16) | (green << 8) | blue;
	}

	delete[] output_h;
	hipFree(output_d);

	return 1;
}
