#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <math.h>
#include <stdio.h>
#include <stdlib.h>

#include "cutil_math.h"

// limited version of checkCudaErrors from hip/hip_runtime_api.h in CUDA examples
#define checkCudaErrors(val) cudaCheck( (val), #val, __FILE__, __LINE__ )

void cudaCheck(hipError_t result, char const *const func, const char *const file, int const line)
{
	if (result) {
		const char *errorName, *errorString;

		errorName = hipGetErrorName(result);
		errorString = hipGetErrorString(result);

		// Make sure we call CUDA Device Reset before exiting
		hipDeviceReset();

		system("Pause");

	}
}

#define M_PI 3.14159265359f  
#define width 512  
#define height 384 
#define samples 512

struct Ray {
	float3 origin;
	float3 direction;
	__device__ Ray(float3 o, float3 d) : origin(o), direction(d) {}
};

enum Material { Diffuse, Specular, Refraction };

struct Sphere {

	float radius;
	float3 position, emission, color;
	Material material;

	__device__ float intersect_sphere(const Ray &r) const {

		float3 op = r.origin - position;
		float t, epsilon = 0.0001f;           // epsilon required to prevent floating point precision artefacts
		float b = dot(op, r.direction);       // b in quadratic equation
											  // discriminant quadratic equation
		float discriminant = b * b - dot(op, op) + radius * radius;
		                                      // if (disc < 0) no real solution (not interested in complex roots) 
		if (discriminant < 0)  return 0;      // else check for solutions using negative and positive discriminant
		else discriminant = sqrtf(discriminant);

		t = -b - discriminant;                // pick closest point in front of ray origin
		if (t > epsilon)	return t;
		else	{
			t = -b + discriminant;
			if (t > epsilon)	return t;
			else				return 0;
		}
	}
};

// Scene
__constant__ Sphere spheres[] = {
	{ 1e5f,{ 1e5f + 1.0f,   40.8f, 81.6f },{ 0.0f, 0.0f, 0.0f },{ 0.75f, 0.25f, 0.25f }, Diffuse },   // Left 
	{ 1e5f,{ -1e5f + 99.0f, 40.8f, 81.6f },{ 0.0f, 0.0f, 0.0f },{ 0.25f, 0.25f, 0.75f }, Diffuse },   // Right 
	{ 1e5f,{         50.0f, 40.8f,  1e5f },{ 0.0f, 0.0f, 0.0f },{ 0.75f, 0.75f, 0.75f }, Diffuse },   // Back 
	{ 1e5f,{ 50.0f, 40.8f, -1e5f + 600.0f},{ 0.0f, 0.0f, 0.0f },{ 1.00f, 1.00f, 1.00f }, Diffuse },   // Front 
	{ 1e5f,{ 50.0f,  1e5f,         81.6f },{ 0.0f, 0.0f, 0.0f },{ 0.75f, 0.75f, 0.75f }, Diffuse },   // Bottom 
	{ 1e5f,{ 50.0f, -1e5f + 81.6f, 81.6f },{ 0.0f, 0.0f, 0.0f },{ 0.75f, 0.75f, 0.75f }, Diffuse },   //Top 
	{ 16.5f,{ 27.0f, 16.5f, 47.0f },{ 0.0f, 0.0f, 0.0f },{ 1.0f, 1.0f, 1.0f },   Specular }, // small glass sphere 1
	{ 16.5f,{ 73.0f, 16.5f, 78.0f },{ 0.0f, 0.0f, 0.0f },{ 1.0f, 1.0f, 1.0f }, Refraction }, // small glass sphere 2
    { 10.5f,{ 50.0f, 46.5f, 90.0f}, { 0.0f, 0.0f, 0.0f },{ 1.0f, 1.0f, 1.0f }, Diffuse },    // small white sphere 3
	{ 600.0f,{ 50.0f, 681.6f - .77f, 81.6f },{ 2.0f, 1.8f, 1.6f },{ 0.0f, 0.0f, 0.0f }, Diffuse }  // Light
};

__device__ inline bool intersect_scene(const Ray &r, float &t, int &id) 
{

	float n = sizeof(spheres) / sizeof(Sphere), d, inf = t = 1e20;

	for (int i = int(n); i--;)
		if ((d = spheres[i].intersect_sphere(r)) && d < t) {
			t = d;
			id = i;
		}
	// returns true if an intersection with the scene occurred, false when no hit
	return t < inf;
}

__device__ static float getrandom(unsigned int *seed0, unsigned int *seed1) 
{
	*seed0 = 36969 * ((*seed0) & 65535) + ((*seed0) >> 16);
	*seed1 = 18000 * ((*seed1) & 65535) + ((*seed1) >> 16);

	unsigned int ires = ((*seed0) << 16) + (*seed1);

	union {
		float f;
		unsigned int ui;
	} res;

	res.ui = (ires & 0x007fffff) | 0x40000000;

	return (res.f - 2.f) / 2.f;
}

__device__ inline void maskWeight(float3& mask, float3 color, float cosine, float scale)
{
	mask *= color;
	mask *= cosine;
	mask *= 2;
}

__device__ float3 loopRadiance(Ray &r, unsigned int *s1, unsigned int *s2)
{
	int depth = 0;
	float3 mask = make_float3(1.0f, 1.0f, 1.0f);
	float3 accucolor = make_float3(0.0f, 0.0f, 0.0f);

	// ray bounce loop 8 Times
	while (depth++ < 8) {
		float t;
		int id = 0;

		// find closest intersection with object's index, if miss break.
		if (!intersect_scene(r, t, id))	break;

		const Sphere &obj = spheres[id];
		float3 x = r.origin + r.direction*t;                 // hitpoint
		float3 n = normalize(x - obj.position);              // normal
		float3 nl = dot(n, r.direction) < 0 ? n : n * -1;    // front facing normal
														 
		r.origin = x + nl * 0.05f;                           // prevent self-intersection	
		accucolor += mask * obj.emission;                    // emissive

		if (obj.material == Diffuse) {    //diffuse
            // uniform sampling hemisphere
			float r1 = 2 * M_PI * getrandom(s1, s2);
			float r2 = getrandom(s1, s2);
			float r2s = sqrtf(r2);

			// compute local coordinate on the hit point
			float3 w = nl;
			float3 u = normalize(cross((fabs(w.x) > .1 ? make_float3(0, 1, 0) : make_float3(1, 0, 0)), w));
			float3 v = cross(w, u);

			float3 d = normalize(u*cos(r1)*r2s + v * sin(r1)*r2s + w * sqrtf(1 - r2));

			r.direction = d;
			r.origin = x + nl * 0.05f;              //offset for self intersection
            // weigh light contribution using cosine of angle between
			// incident light and normal fudge factor
			maskWeight(mask, obj.color, dot(d, nl), 2);
		} else if (obj.material == Specular) {  //specular
			r.direction = r.direction - n * 2 * dot(n, r.direction);
			r.origin = x + r.direction * 0.07f;

			maskWeight(mask, obj.color, dot(r.direction, nl), 2);
		} else { //refraction
			double n1, n2, n3;
			double cosI = dot(n, r.direction);
			if (cosI > 0.0) {
				n1 = 1.5;	n2 = 1.0;	n = -n;
			} else {
				n1 = 1.0;	n2 = 1.5;	cosI = -cosI;
			}
			n3 = n1 / n2;
			double sinT2 = n3 * n3*(1.0 - cosI * cosI);
			double cosT = sqrt(1.0 - sinT2);
			//fernesel equations
			double rn = (n1*cosI - n2 * cosT) / (n1*cosI + n2 * cosT);
			double rt = (n2*cosI - n1 * cosT) / (n2*cosI + n2 * cosT);
			rn *= rn;
			rt *= rt;
			double refl = (rn + rt)*0.5;
			double trans = 1.0 - refl;
			if (n3 == 1.0) {
				maskWeight(mask, obj.color, dot(r.direction, nl), 2);
			}
			//total internal reflection
			if (cosT*cosT < 0.0) {
				r.origin = x + nl * 0.07;
				r.direction = r.direction - n * 2 * dot(n, r.direction);

				maskWeight(mask, obj.color, dot(r.direction, nl), 2);
			}
			else { //refracton
				r.direction = n3 * r.direction + (n3*cosI - cosT)*n;
				r.origin = x + r.direction * 0.07;
				maskWeight(mask, obj.color, dot(r.direction, nl), 2);
			}
		}
	}

	return accucolor;
}

inline  __host__ __device__ float clamp(float x) { return x < 0.0f ? 0.0f : x > 1.0f ? 1.0f : x; }

inline  __host__ __device__ int   toInt(float x) { return int(pow(clamp(x), 1 / 2.2) * 255 + 0.5); }

__global__ void smallptRenderkernel(float3 *output, unsigned int outputSize)
{
	unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
	unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;

	if ((x >= width) || (y >= height)) return;

	unsigned int i = (height - y - 1)*width + x;

	if (i >= outputSize) return;

	unsigned int s1 = x;
	unsigned int s2 = y;

	float3 r = make_float3(0.0f);
	Ray cam(make_float3(50.0, 52.0, 295.6), normalize(make_float3(0.0, -0.042612, -1)));
	float3 cx = make_float3(width * 0.5135 / height, 0.0f, 0.0f);
	float3 cy = normalize(cross(cx, cam.direction)) * 0.5135;

	for (int s = 0; s < samples; s++) {

		float3 d = cam.direction + cx * ((0.25 + x) / width - 0.5) + cy * ((0.25 + y) / height - 0.5);

		r = r + loopRadiance(Ray(cam.origin + d * 40, normalize(d)), &s1, &s2)*(1.0 / samples);
	}

	output[i] = make_float3(clamp(r.x, 0.0f, 1.0f), clamp(r.y, 0.0f, 1.0f), clamp(r.z, 0.0f, 1.0f));
}

extern "C"  int smallpt1RayTrace(unsigned int *sample)
{

	// get number of SMs on this GPU
//	int devID = 0;
//	hipDeviceProp_t deviceProps;
//	checkCudaErrors(hipGetDeviceProperties(&deviceProps, devID));

	hipDeviceReset();

	hipError_t cudaStatus;
	// Choose which GPU to run on
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) return 0;

	float3* output_d;
	float3* output_h = new float3[width*height];
	
	// Allocate GPU buffers for outout image
	cudaStatus = hipMalloc(&output_d, width * height * sizeof(float3));
	if (cudaStatus != hipSuccess) return 0;

	dim3 block(8, 8, 1);
	dim3 grid(width / block.x, height / block.y, 1);

	// Launch a kernel on the GPU with one thread for each element.
	smallptRenderkernel<<< grid, block >>>(output_d, width * height);

	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) return 0;

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) return 0;

	// Copy output vector from GPU buffer to host memory.
	cudaStatus = hipMemcpy(output_h, output_d, width * height * sizeof(float3), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) return 0;

	for (int i = 0; i < width*height; i++) {
		unsigned int red = 0;
		unsigned int green = 0;
		unsigned int blue = 0;

		  red   = toInt(output_h[i].x);
		  green = toInt(output_h[i].y);
		  blue  = toInt(output_h[i].z);

		  sample[i] = (0xFF << 24) | (red << 16) | (green << 8) | blue;
	}

	delete[] output_h;
	hipFree(output_d);

	return 1;
}
